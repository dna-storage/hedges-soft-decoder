#include "hip/hip_runtime.h"
__device__ __forceinline__ FLOAT max3(FLOAT a, FLOAT a1, FLOAT a2) {
    FLOAT maxa = a > a1 ? a : a1; 
    return maxa > a2 ? maxa : a2;
}

__device__ __forceinline__ FLOAT max2(FLOAT a, FLOAT a1) {
    return a > a1 ? a : a1; 
}

__device__ __forceinline__ FLOAT logsumexp3(FLOAT a, FLOAT a1, FLOAT a2) {
    FLOAT maxa = max3(a, a1, a2); 
    return maxa + log(exp(a-maxa) + exp(a1-maxa) + exp(a2-maxa));
}

__device__ __forceinline__ FLOAT logsumexp2(FLOAT a, FLOAT a1) {
  FLOAT maxa = max2(a, a1); 
  return maxa + log(exp(a-maxa) + exp(a1-maxa));
}


__device__ __forceinline__ FLOAT sum3(FLOAT a, FLOAT a1, FLOAT a2) {return a + a1 + a2;}
__device__ __forceinline__ FLOAT add(FLOAT a, FLOAT b) {return a + b;}
__device__ __forceinline__ FLOAT mul(FLOAT a, FLOAT b) {return a * b;}

extern "C" __global__ void fwd_logspace(
					    const FLOAT* __restrict__ target_scores,
					    FLOAT* __restrict__ alpha_t,
					    const FLOAT* __restrict__ mask,
					    const FLOAT* __restrict__ F,
					    int lower_t_range,
					    int upper_t_range,
					    int H,
					    int E,
					    int L,
					    int T,
					    int L_pad,
					    int target_score_pad,
					    int F_offset,
					    int F_T
					)



{
  int Lidx = threadIdx.x, Eidx = threadIdx.z, Hidx=threadIdx.y+blockIdx.x*blockDim.y , blockHidx=threadIdx.y;
  int total_L = L+L_pad;
  int HEL_stride = blockDim.y*E*total_L;
  int EL_stride = E*total_L;
  extern __shared__ FLOAT smem[];
  if(Hidx>=H) return; //get rid of dead threads
  //smem needs to be initialized to time -1 so forward algorithm can go ahead
  if(Lidx==0) smem[(lower_t_range%2)*HEL_stride+ blockHidx*EL_stride + Eidx*(total_L) + Lidx] = ZERO;
  if(Lidx==1) smem[(lower_t_range%2)*HEL_stride+ blockHidx*EL_stride + Eidx*(total_L) + Lidx] = F[(lower_t_range+F_offset)*H+Hidx];
  smem[(lower_t_range%2)*HEL_stride+blockHidx*EL_stride+Eidx*(total_L)+(Lidx+L_pad)] = ZERO;
  __syncthreads();
  for(int t=lower_t_range; t<upper_t_range;t++){
    //perform core calculations for forward algorithm
    FLOAT a,a1,a2,final_score,score; //a->current string step, a1-> one string step back, a2->two string steps back
    a = smem[(t%2)*HEL_stride+ blockHidx*EL_stride+ Eidx*total_L+ (Lidx+L_pad)];
    a1 = smem[(t%2)*HEL_stride+ blockHidx*EL_stride+ Eidx*total_L + (Lidx+L_pad-1)];
    a2 =  MUL(smem[(t%2)*HEL_stride + blockHidx*EL_stride + Eidx*total_L+ (Lidx+L_pad-2)],mask[Hidx*E*L+Eidx*L+Lidx]);
    score = target_scores[t*H*E*(L+target_score_pad)+ Hidx*E*(L+target_score_pad)+ Eidx*(L+target_score_pad)+ (Lidx+target_score_pad)];
    //if(Hidx==0 && Eidx==0) printf("t %d L %d score %f \n",t,Lidx,score);
    final_score = MUL(score,SUM(a,a1,a2));
    smem[(((t+1)%2))*HEL_stride + blockHidx*EL_stride + Eidx*total_L+(Lidx+L_pad)]=final_score;
    if(Lidx==0) smem[(((t+1)%2))*HEL_stride+ blockHidx*EL_stride+ Eidx*(total_L)+ Lidx] = ZERO;
    else if (Lidx==1){
      int f_t = (t+1+F_offset);
      if(f_t<F_T) smem[((t+1)%2)*HEL_stride + blockHidx*EL_stride + Eidx*total_L + Lidx] = F[f_t*H + Hidx];
      else smem[((t+1)%2)*HEL_stride + blockHidx*EL_stride + Eidx*total_L + Lidx] = ZERO;
    }
    if (Lidx==L-1) alpha_t[ t*H*E+ Hidx*E+ Eidx] = final_score;
    __syncthreads();
  }
}





extern "C" __global__ void fwd_logspac_align(
					    const FLOAT* __restrict__ target_scores,
					    FLOAT* __restrict__ F,
					    FLOAT* __restrict__ BT,
					    const FLOAT* __restrict__ mask,
					    int T,
					    int L,
					    int L_pad
					)



{
  int Lidx = threadIdx.x;
  int total_L = L+L_pad;
  extern __shared__ FLOAT smem[];
  if(Lidx==0 || Lidx==1) smem[(lower_t_range%2)*total_L+Lidx] = ZERO;
  smem[(lower_t_range%2)*total_L+(Lidx+L_pad)] = ZERO;
  __syncthreads();
  for(int t=0 t<T;t++){
    //perform core calculations for forward algorithm
    FLOAT a,a1,a2,final_score,score; //a->current string step, a1-> one string step back, a2->two string steps back
    score = target_scores[t*(L)+Lidx];
    a = MUL(score,smem[(t%2)*total_L+(Lidx+L_pad)]);
    a1 = MUL(score,smem[(t%2)*total_L+(Lidx+L_pad-1)]);
    a2 =  MUL(score,smem[(t%2)*total_L+(Lidx+L_pad-2)],mask[Lidx]);
    final_score = max3(a,a1,a2);
    int a_ = (a>a1 && a>a2)*0;
    int a1_ = (a1>a && a1>a2)*1;
    int a2_ = (a2>a && a2>a1)*2; 
    F[(t*L+Lidx)]=final_score;
    if(Lidx>0) BT[t*(L-1)+Lidx-1)]=Lidx-(a_+a1_+a2_);
    smem[(((t+1)%2))*total_L+(Lidx+L_pad)]=final_score;
    if(Lidx==0 || Lidx==1) smem[(((t+1)%2))*total_L+Lidx] = ZERO;
    __syncthreads();
  }
}

//this is an optimized verision of fwd_logspace, seems like it may not be most important to use right now
extern "C" __global__ void fwd_logspace_opt(
					    const FLOAT* __restrict__ target_scores,
					    FLOAT* __restrict__ alpha_t,
					    const FLOAT* __restrict__ mask,
					    const FLOAT* __restrict__ F,
					    int lower_t_range,
					    int upper_t_range,
					    int H,
					    int E,
					    int L,
					    int T,
					    int L_pad)



{
  int tid = threadIdx.x;
  int smem_tid;
  int total_L = L+L_pad;
  int HEL_stride = H*E*total_L;
  extern __shared__ FLOAT smem[];
  FLOAT* buff_0 = smem;
  FLOAT* buff_1 = smem+HEL_stride;
  int Hidx = tid/(L*E);
  int Eidx = (tid/L)-Hidx*E;
  int Lidx = tid - (Eidx*L) - (Hidx*L*E);
  if (Lidx==0) smem_tid=tid*2;
  else smem_tid=tid*2-1;
  //smem needs to be initialized to time -1 so forward algorithm can go ahead
  buff_0[smem_tid] = F[(lower_t_range-1)*H*L+Hidx*L_pad+Lidx];
  buff_0[smem_tid+L_pad] = ZERO;
  __syncthreads();
  for(int t=lower_t_range; t<upper_t_range;t++){
    FLOAT* temp_buff;
    //perform core calculations for forward algorithm
    FLOAT a,a1,a2,final_score,score; //a->current string step, a1-> one string step back, a2->two string steps back
    a = buff_0[smem_tid+L_pad];
    a1 = buff_0[smem_tid+L_pad-1];
    a2 =  MUL(buff_0[smem_tid],mask[tid]);
    score = target_scores[t*H*E*L+tid];
    final_score = MUL(score,SUM(a,a1,a2));
    buff_1[smem_tid+L_pad]=final_score;
    buff_1[smem_tid] = F[t*H*L_pad + Hidx*L_pad+ Lidx];
    if(Lidx==1) alpha_t[t*H*E+Hidx*E+Eidx] = final_score;
    //rotate buffers
    temp_buff=buff_0;
    buff_0=buff_1;
    buff_1=buff_0;
    __syncthreads();
  }
}

extern "C" __global__ void dot_mul(
					    const FLOAT* __restrict__ target_scores,
					    const FLOAT* __restrict__ alpha_t,
					    FLOAT* __restrict__ output,
					    int T,
					    int target_scores_L
				   )
{
  int Hidx = threadIdx.y;
  int Eidx = threadIdx.x;
  int Tidx = threadIdx.z+blockDim.z*blockIdx.x;
  if(Tidx<T){
    int H = blockDim.y;
    int E = blockDim.x;
    int idx = Tidx*H*E+Hidx*E+Eidx;
    int idx2= (idx+H*E)*target_scores_L+target_scores_L-1;
    if(Tidx+1<T)
      output[idx] = MUL(alpha_t[idx],SUM2(0,target_scores[idx2]));
    else
      output[idx] = alpha_t[idx];
  }
}

